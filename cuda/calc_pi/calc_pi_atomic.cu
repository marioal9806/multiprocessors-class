// nvcc -arch=sm_60 calc_pi.cu -o calc_pi


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

clock_t start, end;

__global__ void calc_pi_kernel(long n, double* acum) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    
    double x = 0;
    double fdx = 0;
    double baseIntervalo = 1.0 / n;

    double localAcum = 0;

    for (long i = index; i < n; i += stride) {
        x = (i+0.5)*baseIntervalo;
        fdx = 4 / (1 + x * x);
        localAcum += fdx;
    }
    atomicAdd(acum, localAcum);
}

int main() {
    
    double *acum;
    int size = sizeof(double);

    long cantidadIntervalos = 1000000000;
    double baseIntervalo = 1.0 / cantidadIntervalos;

    // Definir el numero de threads a utilizar en el device
    int blockSize = 1024;
    int numBlocks = 16384;

    // Asignar un acumulador local para cada thread
    int ret = 0;
    ret = hipMallocManaged((void **)&acum, size);
    if (ret) {
        printf("Error on CUDA Malloc: %d\n", ret);
    }
    
    hipMemset(acum, 0, size);

    start = clock();
    calc_pi_kernel<<<numBlocks, blockSize>>>(cantidadIntervalos, acum);
    hipDeviceSynchronize();
    end = clock();

    *acum *= baseIntervalo; 

    printf("Result = %20.18lf (%ld)\n", *acum, end - start);

    hipFree(acum);

    return 0;
}