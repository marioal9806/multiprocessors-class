// nvcc calc_pi.cu -o calc_pi


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

clock_t start, end;

__global__ void calc_pi_kernel(long n, double* acum) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    
    double x = 0;
    double fdx = 0;
    double baseIntervalo = 1.0 / n;

    for (long i = index; i < n; i += stride) {
        x = (i+0.5)*baseIntervalo;
        fdx = 4 / (1 + x * x);
        acum[index] += fdx;
    }
}

int main() {
    
    double *d_acum;
    double* h_acum;
    int size = sizeof(double);

    long h_cantidadIntervalos = 1000000000;
    double baseIntervalo = 1.0 / h_cantidadIntervalos;

    // Definir el numero de threads a utilizar en el device
    int blockSize = 1024;
    int numBlocks = 16384;
    int numThreads = blockSize*numBlocks;

    // Asignar un acumulador local para cada thread
    int ret = 0;
    ret = hipMalloc((void **)&d_acum, size*numThreads);
    if (ret) {
        printf("Error on CUDA Malloc: %d\n", ret);
    }
    
    h_acum = (double*)malloc(size*numThreads);
    if (h_acum == NULL) {
        printf("Error on malloc\n");
    }

    start = clock();
    calc_pi_kernel<<<numBlocks, blockSize>>>(h_cantidadIntervalos, d_acum);
    hipDeviceSynchronize();
    end = clock();

    // Get individual results back from device
    hipMemcpy(h_acum, d_acum, size*numThreads, hipMemcpyDeviceToHost);
    
    // Reduce final result on the host
    double acum_total = 0;
    for (int i = 0; i < numThreads; i++) {
        acum_total += h_acum[i];
    }
    acum_total *= baseIntervalo;

    printf("Result = %20.18lf (%ld)\n", acum_total, end - start);

    hipFree(d_acum);
    free(h_acum);

    return 0;
}