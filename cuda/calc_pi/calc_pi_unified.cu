// nvcc calc_pi.cu -o calc_pi


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

clock_t start, end;

__global__ void calc_pi_kernel(long n, double* acum) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    
    double x = 0;
    double fdx = 0;
    double baseIntervalo = 1.0 / n;

    for (long i = index; i < n; i += stride) {
        x = (i+0.5)*baseIntervalo;
        fdx = 4 / (1 + x * x);
        acum[index] += fdx;
    }
}

int main() {
    
    double *acum;
    int size = sizeof(double);

    long cantidadIntervalos = 1000000000;
    double baseIntervalo = 1.0 / cantidadIntervalos;

    // Definir el numero de threads a utilizar en el device
    int blockSize = 1024;
    int numBlocks = 16384;
    int numThreads = blockSize*numBlocks;

    // Asignar un acumulador local para cada thread
    int ret = 0;
    ret = hipMallocManaged((void **)&acum, size*numThreads);
    if (ret) {
        printf("Error on CUDA Malloc: %d\n", ret);
    }
    
    hipMemset(acum, 0, size*numThreads);

    start = clock();
    calc_pi_kernel<<<numBlocks, blockSize>>>(cantidadIntervalos, acum);
    hipDeviceSynchronize();
    end = clock();

    
    // Reduce final result on the host
    double acum_total = 0;
    for (int i = 0; i < numThreads; i++) {
        acum_total += acum[i];
    }
    acum_total *= baseIntervalo;

    printf("Result = %20.18lf (%ld)\n", acum_total, end - start);

    hipFree(acum);

    return 0;
}