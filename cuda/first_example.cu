
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void c_hello() {
    printf("Hello from cuda!\n");
}

int main(int argc, char const *argv[])
{
    c_hello<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
